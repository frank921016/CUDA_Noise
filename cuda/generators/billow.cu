#include "hip/hip_runtime.h"
#include "billow.cuh"


__device__ float billow2D(float2 point, hipTextureObject_t perm, hipTextureObject_t grad, float freq, float lacun, float persist, int init_seed, int octaves) {
	// Will be incremented upon.
	float result = 0.0f;
	float amplitude = 0.95f;
	float val;
	// Scale point by freq
	point.x = point.x * freq;
	point.y = point.y * freq;
	// TODO: Seeding the function is currently pointless and doesn't actually do anything.
	// Use loop for octav-ing
	for (size_t i = 0; i < octaves; ++i) {
		int seed = (init_seed + i) & 0xffff;
		val = perlin2d(perm, grad, point, seed);
		val = fabsf(val);
		result += val * amplitude;
		// Modify vars for next octave.
		freq *= lacun;
		point.x *= freq;
		point.y *= freq;
		amplitude *= persist;
	}
	float tmp = result / 100.0f;
	// * // 
	return tmp;
}

__global__ void Billow2DKernel(hipSurfaceObject_t out, hipTextureObject_t perm, hipTextureObject_t grad, int width, int height, float2 origin, float freq, float lacun, float persist, int seed, int octaves) {
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i >= width || j >= height) {
		return;
	}

	float x, y;
	x = i + origin.x;
	y = j + origin.y;
	float2 p = make_float2(x, y);
	// Call billow function
	float val = billow2D(p, perm, grad, freq, lacun, persist, seed, octaves);

	// Write val to the surface
	surf2Dwrite(val, out, i * sizeof(float), j);
}


void BillowLauncher(hipSurfaceObject_t out, hipTextureObject_t perm, hipTextureObject_t grad, int width, int height, float2 origin, float freq, float lacun, float persist, int seed, int octaves) {

#ifdef CUDA_TIMING_TESTS
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
#endif // CUDA_TIMING_TESTS

	// Setup dimensions of kernel launch. 
	
	// Use occupancy calc to find optimal sizes.
	int blockSize, minGridSize;
#ifdef CUDA_TIMING_TESTS
	hipEventRecord(start);
#endif // CUDA_TIMING_TESTS
	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, (void*)Billow2DKernel, 0, 0);
	dim3 block(blockSize, blockSize, 1);
	dim3 grid((width - 1) / blockSize + 1, (height - 1) / blockSize + 1, 1);
	Billow2DKernel<<<block,grid>>>(out, perm, grad, width, height, origin, freq, lacun, persist, seed, octaves);
	// Check for succesfull kernel launch
	cudaAssert(hipGetLastError());
	// Synchronize device
	cudaAssert(hipDeviceSynchronize());
#ifdef CUDA_TIMING_TESTS
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float elapsed = 0.0f;
	hipEventElapsedTime(&elapsed, start, stop);
	printf("Kernel execution time in ms: %f\n", elapsed);
#endif // CUDA_TIMING_TESTS
	// If this completes, kernel is done and "output" contains correct data.
}