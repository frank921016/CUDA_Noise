#include "hip/hip_runtime.h"
#include "billow.cuh"


__device__ float billow2D(float2 point, hipTextureObject_t perm, float freq, float lacun, float persist, int init_seed, int octaves) {
	// Will be incremented upon.
	float result = 0.0f;
	float val = 0.0f;
	float curPersistence = 1.0f;
	// Scale point by freq
	point.x *= freq;
	point.y *= freq;
	// TODO: Seeding the function is currently pointless and doesn't actually do anything.
	// Use loop for octav-ing
	for (size_t i = 0; i < octaves; ++i) {
		//int seed = (init_seed + i) & 0xffffffff;
		val = perlin2d(point, perm);
		val = 2.0f * fabsf(val) - 1.0f;
		result += val * curPersistence;
		// Modify vars for next octave.
		point.x *= lacun;
		point.y *= lacun;
		curPersistence *= persist;
	}

	result += 0.50f;
	return result;
}

__global__ void Billow2DKernel(hipSurfaceObject_t out, hipTextureObject_t perm, int width, int height, float2 origin, float freq, float lacun, float persist, int seed, int octaves) {
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i >= width || j >= height) {
		return;
	}

	float x, y;
	x = i + origin.x;
	y = j + origin.y;
	float2 p = make_float2(x, y);
	// Call billow function
	float val = billow2D(p, perm, freq, lacun, persist, seed, octaves);

	// Write val to the surface
	surf2Dwrite(val, out, i * sizeof(float), j);
}


void BillowLauncher(hipSurfaceObject_t out, hipTextureObject_t perm, int width, int height, float2 origin, float freq, float lacun, float persist, int seed, int octaves) {
	// Setup dimensions of kernel launch. 
	
	// Use occupancy calc to find optimal sizes.
	int blockSize, minGridSize;
	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, (void*)Billow2DKernel, 0, 0);
	dim3 block(blockSize, blockSize, 1);
	dim3 grid((width - 1) / blockSize + 1, (height - 1) / blockSize + 1, 1);
	if (grid.x > static_cast<unsigned int>(minGridSize) || grid.y > static_cast<unsigned int>(minGridSize)) {
		throw("Grid sizing error.");
	}
	Billow2DKernel<<<block,grid>>>(out, perm, width, height, origin, freq, lacun, persist, seed, octaves);

	// Check for succesfull kernel launch
	cudaAssert(hipGetLastError());

	// Synchronize device
	cudaAssert(hipDeviceSynchronize());

	// If this completes, kernel is done and "output" contains correct data.
}